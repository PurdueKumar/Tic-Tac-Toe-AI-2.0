#include "hip/hip_runtime.h"

#include "ai.cuh"
#include <fstream>
#include <string.h>
#include <cmath>

using namespace std;

ai::ai(int i, int h, int o, int pnum) {

	hipError_t cudaStatus;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&handle);

	this->il = i;
	this->hl = h;
	this->ol = o;
	this->rcount = 0;
	this->pnum = pnum;

	ifstream theta1;
	ifstream theta2;

	theta1.open("Theta1win.csv");
	theta2.open("Theta2win.csv");

	cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto cleanup;
	}

	cudaStatus = hipMalloc((void**)&t1, (sizeof(double) * h * (i + 1)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for t1 failed!");
		goto cleanup;
	}

	cudaStatus = hipMalloc((void**)&t2, (sizeof(double) * (h+i) * o));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for t2 failed!");
		goto cleanup;
	}

	cudaStatus = hipMalloc((void**)&pred, (sizeof(double) * 9));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for pred failed!");
		goto cleanup;
	}

	if (theta1.fail() || theta2.fail()) {

		cout << "failed to open theta file\n";
		system("pause");
		exit(1);

	}

	int t1num = populateMatrix(theta1, t1, h * (i + 1));
	int t2num = populateMatrix(theta2, t2, (h + i) * o);

	theta1.close();
	theta2.close();

cleanup:

	hipFree(t1);
	hipFree(t2);
	theta1.close();
	theta2.close();
}

__global__ void sigKernal(double * t) {

	int i = threadIdx.x;
	t[i] = 1 / (1 + exp(-1 * t[i]));

}

void ai::sigmoid(double * t, int n) {

	hipError_t cudaStatus;

	sigKernal <<<1, n >>> (t);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

	return;

}

int ai::populateMatrix(ifstream& in, double * t, int n) {

	char * line;
	char * tok;
	char * next_token;
	double num;
	int i = 0;
	double * temp = (double *)malloc(sizeof(double) * n);
	hipError_t cudaStatus;

	line = (char *)malloc(sizeof(char) * 1000);

	while (1) {

		in.getline(line, 1000);
		if (in.eof()) {

			break;

		}

		tok = strtok_s(line, ",", &next_token);

		while (tok) {

			num = atof(tok);
			temp[i] = num;
			i++;

			tok = strtok_s(NULL, ",", &next_token);

		}

	}
	free(line);

	cudaStatus = hipMemcpy(t, temp, n, hipMemcpyHostToDevice);

	free(temp);

	return i;
}

void ai::reverseGrid(int * grid) {

	if (this->pnum == 1) {

		return;

	}

	for (int i = 0; i < 9; i++) {

		if (grid[i] == 1) {

			grid[i] = 2;

		}

		else if (grid[i] == 2) {

			grid[i] = 1;

		}

	}

}

void ai::multiply(double * a, double * b, double * c, int m, int k, int n) {

	hipblasStatus_t hipblasStatus_t;
	const double coeff = 1;

	hipblasStatus_t = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &coeff, a, k, b, n, &coeff, c, n);

}

int ai::move(int * pMoves, int * grid) {

	hipError_t cudaStatus;
	hipblasStatus_t hipblasStatus_t;
	double * input;
	double * hunits;

	int bias = 1;

	reverseGrid(grid);

	cudaStatus = hipMalloc((void**)&input, (sizeof(double) * 10));
	cudaStatus = hipMalloc((void**)&hunits, (sizeof(double) * 51));

	hipMemcpy(&input[0], &bias, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(&hunits[0], &bias, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(&input[1], &grid, sizeof(int) * 9, hipMemcpyHostToDevice);

	multiply(t1, input, &hunits[1], 50, 10, 1);
	sigmoid(hunits, 51);

	multiply(t2, hunits, pred, 9, 51, 1);
	sigmoid(pred, 9);

	int m;
	hipblasStatus_t = hipblasIdamax(handle, 9, pred, 1 , &m);

	for (int i = 1; i < (pMoves[0] + 1); i++) {

		if (m == pMoves[i]) {

			cout << "AI move is " << m << endl;
			hipFree(input);
			hipFree(hunits);

			return m;
		}

	}

	m = rand() % pMoves[0] + 1;
	cout << "random move is " << pMoves[m] << endl;
	rcount++;

	hipFree(input);
	hipFree(hunits);

	return pMoves[m];

}

void ai::printr() {

	cout << "number of random AI moves: " << this->rcount << endl;
}


ai::~ai() {

	hipFree(t1);
	hipFree(t2);
	hipFree(pred);

	hipblasDestroy(handle);

	
}